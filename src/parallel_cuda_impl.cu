#include "hip/hip_runtime.h"
#include "../inc/parallel_cuda_impl.cuh"

inline void gpuAssert(hipError_t code, char *file, int line, bool abort)
{
    if (code != hipSuccess) 
    {
        // fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__, true); }

//Warning: This mmodifies the nonce value of data so do it last!
void compute_and_print_hash(unsigned char *data, unsigned int nonce)
{
    unsigned char hash[32];
    SHA256_CTX ctx;
    int i;
    
    *((unsigned long *) (data + 76)) = ENDIAN_SWAP_32(nonce);

    sha256_init(&ctx);
    sha256_update(&ctx, data, 80);
    sha256_final(&ctx, hash);
    sha256_init(&ctx);
    sha256_update(&ctx, hash, 32);
    sha256_final(&ctx, hash);

    printf("Hash is:\n");
    for(i=0; i<8; i++) {
        printf("%.8x ", ENDIAN_SWAP_32(*(((unsigned int *) hash) + i)));
    }
    printf("\n");
}

__global__ void kernel_sha256d(SHA256_CTX *ctx, Nonce_result *nr, void *debug)
{
    unsigned int m[64];
    unsigned int hash[8];
    unsigned int arr[8];
    int i, j;
    unsigned int nonce = NONCE_VAL;
    // printf("Enter kernel with nonce_val:%d\n", nonce);

    //Compute SHA-256 Message Schedule
    unsigned int *le_data = (unsigned int *) ctx->data;
    for(i=0; i<16; i++)
    {
        m[i] = le_data[i];
    }
    //Replace the nonce
    m[3] = nonce;
    for ( ; i < 64; ++i)
    {
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
    }

    //Copy Initial Values into registers
    for (i = 0; i < 8; i++)
    {
        arr[i] = ctx->state[i];
    }

    //This is a large multiline macro for the SHA256 compression rounds
    // SHA256_COMPRESS_8X
    sha256_compress_8x_gpu(arr, gpu_k, m);

    //Prepare input for next SHA-256
    for (i = 0; i < 8; i++)
    {
        m[i] = arr[i] + ctx->state[i];
    }
    //Pad the input
    m[8] = 0x80000000;
    for(i=9; i<15; i++)
    {
        m[i] = 0x00;
    }
    m[15] = 0x00000100;    //Write out l=256
    for (i=16 ; i < 64; ++i)
    {
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
    }

    //Initialize the SHA-256 registers
    arr[0] = 0x6a09e667;
    arr[1] = 0xbb67ae85;
    arr[2] = 0x3c6ef372;
    arr[3] = 0xa54ff53a;
    arr[4] = 0x510e527f;
    arr[5] = 0x9b05688c;
    arr[6] = 0x1f83d9ab;
    arr[7] = 0x5be0cd19;

    // SHA256_COMPRESS_1X
    sha256_compress_1x_gpu(arr, gpu_k, m);

    hash[0] = ENDIAN_SWAP_32(arr[0] + 0x6a09e667);
    hash[1] = ENDIAN_SWAP_32(arr[1] + 0xbb67ae85);
    hash[2] = ENDIAN_SWAP_32(arr[2] + 0x3c6ef372);
    hash[3] = ENDIAN_SWAP_32(arr[3] + 0xa54ff53a);
    hash[4] = ENDIAN_SWAP_32(arr[4] + 0x510e527f);
    hash[5] = ENDIAN_SWAP_32(arr[5] + 0x9b05688c);
    hash[6] = ENDIAN_SWAP_32(arr[6] + 0x1f83d9ab);
    hash[7] = ENDIAN_SWAP_32(arr[7] + 0x5be0cd19);

    #ifdef VERIFY_HASH
    unsigned int *ref_hash = (unsigned int *) debug;
    for(i=0; i<8; i++)
    {
        cuPrintf("%.8x, %.8x\n", hash[i], ref_hash[i]);
    }
    #endif

    unsigned char *hhh = (unsigned char *) hash;
    i=0;
    while(hhh[i] == ctx->difficulty[i])
    {
        i++;
    }

    if(hhh[i] < ctx->difficulty[i])
    {
        //Synchronization Issue
        //Kind of a hack but it really doesn't matter which nonce
        //is written to the output, they're all winners :)
        //Further it's unlikely to even find a nonce let alone 2
        nr->nonce_found = true;
        //The nonce here has the correct endianess,
        //but it must be stored in the block in little endian order
        nr->nonce = nonce;
    }
    // printf("Exit kernel.\n");
}

void* parallel_cuda_impl(void* threadArg)
{
    GPUData* gpuDataPtr = (GPUData*) threadArg;
    unsigned int gridDimX = gpuDataPtr->m_gridDimX;
    unsigned int gridDimY = gpuDataPtr->m_gridDimY;
    unsigned int blockDimX = gpuDataPtr->m_blockDimX;
    unsigned int difficulty = gpuDataPtr->m_difficulty;

    int i, j;
    unsigned char* data = test_block;

    /*
        Host Side Preprocessing
        The goal here is to prepare and compute everything that will be shared by all threads.
    */
    
    //Initialize Cuda stuff
    // cudaPrintfInit();
    dim3 DimGrid(gridDimX, gridDimY);
    #ifndef ITERATE_BLOCKS
    dim3 DimBlock(blockDimX, 1);
    #endif

    //Used to store a nonce if a block is mined
    Nonce_result h_nr;
    initialize_nonce_result(&h_nr);

    //Compute the shared portion of the SHA-256d calculation
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, (unsigned char *) data, 80);    //ctx.state contains a-h
    sha256_pad(&ctx);
    //Rearrange endianess of data to optimize device reads
    unsigned int *le_data = (unsigned int *)ctx.data;
    unsigned int le;
    for(i=0, j=0; i<16; i++, j+=4) {
        //Get the data out as big endian
        //Store it as little endian via x86
        //On the device side cast the pointer as int* and dereference it correctly
        le = (ctx.data[j] << 24) | (ctx.data[j + 1] << 16) | (ctx.data[j + 2] << 8) | (ctx.data[j + 3]);
        le_data[i] = le;
    }

    //Decodes and stores the difficulty in a 32-byte array for convenience
    // unsigned int nBits = ENDIAN_SWAP_32(*((unsigned int *) (data + 72)));
    set_difficulty(ctx.difficulty, difficulty);                                //ctx.data contains padded data

    //Data buffer for sending debug information to/from the GPU
    unsigned char debug[32];
    unsigned char *d_debug;
    #ifdef VERIFY_HASH
    SHA256_CTX verify;
    sha256_init(&verify);
    sha256_update(&verify, (unsigned char *) data, 80);
    sha256_final(&verify, debug);
    sha256_init(&verify);
    sha256_update(&verify, (unsigned char *) debug, 32);
    sha256_final(&verify, debug);
    #endif
    CUDA_SAFE_CALL(hipMalloc((void **)&d_debug, 32*sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpy(d_debug, (void *) &debug, 32*sizeof(unsigned char), hipMemcpyHostToDevice));

    //Allocate space on Global Memory
    SHA256_CTX *d_ctx;
    Nonce_result *d_nr;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_ctx, sizeof(SHA256_CTX)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));

    /*
        Kernel Execution
        Measure and launch the kernel and start mining
    */
    //Copy data to device
    CUDA_SAFE_CALL(hipMemcpy(d_ctx, (void *) &ctx, sizeof(SHA256_CTX), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_nr, (void *) &h_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));

    float elapsed_gpu;
    long long int num_hashes;
    #ifdef ITERATE_BLOCKS
    //Try different block sizes
    for(i=1; i <= 512; i++) {
        dim3 DimBlock(i,1);
    #endif
        //Start timers
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        //Launch Kernel
        kernel_sha256d<<<DimGrid, DimBlock>>>(d_ctx, d_nr, (void *) d_debug);

        //Stop timers
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_gpu, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

    #ifdef ITERATE_BLOCKS
        //Calculate results
        num_hashes = GDIMX*i;
        //block size, hashrate, hashes, execution time
        printf("%d, %.2f, %.0f, %.2f\n", i, num_hashes/(elapsed_gpu*1e-3), num_hashes, elapsed_gpu);
    }
    #endif
    //Copy nonce result back to host
    CUDA_SAFE_CALL(hipMemcpy((void *) &h_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));

    /*    
        Post Processing
        Check the results of mining and print out debug information
    */

    //Cuda Printf output
    hipDeviceSynchronize();
    // cudaPrintfDisplay(stdout, true);
    // cudaPrintfEnd();

    //Free memory on device
    CUDA_SAFE_CALL(hipFree(d_ctx));
    CUDA_SAFE_CALL(hipFree(d_nr));
    CUDA_SAFE_CALL(hipFree(d_debug));

    //Output the results
    if(h_nr.nonce_found) {
        printf("[CUDA] Nonce found! %.8x\n", h_nr.nonce);
        // compute_and_print_hash(data, h_nr.nonce);
    }
    else {
        printf("[CUDA] Nonce not found :(\n");
    }
    
    num_hashes = blockDimX * gridDimX * gridDimY;
    printf("[CUDA] Tested %lld hashes\n", num_hashes);
    printf("[CUDA] GPU execution time: %f ms\n", elapsed_gpu);
    printf("[CUDA] Hashrate: %.2f H/s\n", num_hashes/(elapsed_gpu*1e-3));

    gpuDataPtr->m_time = elapsed_gpu * 1e-3;

    return 0;
}

__device__ void sha256_compress_1x_gpu(unsigned int* arr, const unsigned int* k, unsigned int* m)
{
    unsigned int t1;
    unsigned int t2;
    int i, j;
    for (i = 0; i < 64; i++)
    {
        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i] + m[i];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }
    }
}

__device__ void sha256_compress_8x_gpu(unsigned int* arr, const unsigned int* k, unsigned int* m)
{
    unsigned int t1;
    unsigned int t2;
    int i, j;
    for (i = 0; i < 64; i += 8)
    {
        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i] + m[i];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 1] + m[i + 1];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 2] + m[i + 2];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 3] + m[i + 3];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 4] + m[i + 4];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 5] + m[i + 5];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 6] + m[i + 6];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }

        t1 = arr[7] + EP1(arr[4]) + CH(arr[4], arr[5], arr[6]) + k[i + 7] + m[i + 7];
        t2 = EP0(arr[0]) + MAJ(arr[0], arr[1], arr[2]);
        for (j = 7; j >= 0; j--)
        {
            if (j == 4)
            {
                arr[j] = arr[j - 1] + t1;
            }
            else if (j == 0)
            {
                arr[j] = t1 + t2;
            }
            else
            {
                arr[j] = arr[j - 1];
            }
        }
    }
}
