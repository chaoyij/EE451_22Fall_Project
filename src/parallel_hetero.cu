#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
    #include "../inc/sha256.h"
    #include "../inc/utils.h"
}

#include "../inc/parallel_pthread.h"

// CUDA Stuff
#define BDIMX   64      //Max = 512
#define GRIMX   8192    //MAX = 8192 65536
#define GRIMY   8192
__global__ void kernel_sha256d(SHA256_CTX *ctx, Nonce_result *nr, void *debug, unsigned int *hash_limit);


inline void gpuAssert(hipError_t code, char *file, int line, bool abort)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__, true); }

void compute_and_print_hash(unsigned char *data, unsigned int nonce) {
    unsigned char hash[32];
    SHA256_CTX ctx;
    int i;
                
    *((unsigned long *) (data + 76)) = ENDIAN_SWAP_32(nonce);

    sha256_init(&ctx);
    sha256_update(&ctx, data, 80);
    sha256_final(&ctx, hash);
    sha256_init(&ctx);
    sha256_update(&ctx, hash, 32);
    sha256_final(&ctx, hash);

    printf("Hash is:\n");
    for(i=0; i<8; i++) {
        printf("%.8x ", ENDIAN_SWAP_32(*(((unsigned int *) hash) + i)));
    }
    printf("\n");
}   


int main(int argc, char* argv[])
{
    if (argc < 6)
    {
        printf("Wrong argument. Sample correct format: ./hetero_miner maxNumBits maxDifficultyBits maxNumThreads percentageCPU percentageYes\n");
        return -1;
    }

    const unsigned int maxNumBits = atoi(argv[1]);
    const unsigned int maxDifficultyBits = atoi(argv[2]);
    const unsigned int maxNumThreads = atoi(argv[3]);

    // Percentage on the job distribution
    const unsigned int percentageCPU = atoi(argv[4]);
    const unsigned int percentageYes = atoi(argv[5]);
    int i, j;
    unsigned char* data = test_block;
    unsigned char hash[32];
    

    // Initialize Pthread Implementation
    const unsigned int MaxThreads = maxNumThreads;
    //const unsigned int hashes = (1 << maxNumBits);

    unsigned int hashes;
    if (!percentageYes)
         hashes = (1 << maxNumBits);
    else
    {
        hashes =  (unsigned int) BDIMX*percentageCPU/100;
        hashes *= (unsigned int) GRIMX*GRIMY;
    }
    const unsigned int threadNum = hashes <= MaxThreads ? hashes : MaxThreads;  
    const unsigned int elementsPerThread = hashes / threadNum;
    pthread_t threads[threadNum];
    ThreadData threadDataArray[threadNum];
     
    
    // Initialize CPU CTX
    SHA256_CTX cpu_ctx;
    Nonce_result cpu_nr;
    struct timespec start;
    struct timespec stop; 
    double cpu_time;
    initialize_nonce_result(&cpu_nr);
    sha256_init(&cpu_ctx);
    sha256_update(&cpu_ctx, (unsigned char*) data, 80);    //ctx.state contains a-h
    sha256_pad(&cpu_ctx);
    customize_difficulty(cpu_ctx.difficulty, maxDifficultyBits);
   
    // Initialize CUDA stuff
    dim3 DimGrid(GRIMX,GRIMY);
    dim3 DimBlock(BDIMX,1);
    float gpu_time;
    long long int num_hashes;
    SHA256_CTX gpu_ctx;
    Nonce_result gpu_nr;
    initialize_nonce_result(&gpu_nr);
    sha256_init(&gpu_ctx);
    sha256_update(&gpu_ctx, (unsigned char*) data, 80);
    sha256_pad(&gpu_ctx);
    // unsigned int nBits = ENDIAN_SWAP_32(*((unsigned int *) (data + 72)));
    // set_difficulty(gpu_ctx.difficulty, nBits);
    customize_difficulty(gpu_ctx.difficulty, maxDifficultyBits);
    
    // Data buffer for sending debug information to/from the GPU
    unsigned char debug[32];
    unsigned char *d_debug;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_debug, 32*sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpy(d_debug, (void *) &debug, 32*sizeof(unsigned char), hipMemcpyHostToDevice));
    
    // Allocate Space on Global Memory 
    SHA256_CTX *d_ctx;
    Nonce_result *d_nr;
    unsigned int *d_hashes;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_ctx, sizeof(SHA256_CTX)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));
    // Checkpoint
    CUDA_SAFE_CALL(hipMalloc(&d_hashes, sizeof(int)));

    //Copy data to device
    CUDA_SAFE_CALL(hipMemcpy(d_ctx, (void *) &gpu_ctx, sizeof(SHA256_CTX), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_nr, (void *) &gpu_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));
    // Checkpoint
    CUDA_SAFE_CALL(hipMemcpy(d_hashes, (unsigned int*) &hashes, sizeof(int), hipMemcpyHostToDevice ));

    // Start GPU Timer
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start, 0);
 
    // Start CPU Timer
    if(clock_gettime(CLOCK_REALTIME, &start) == -1)
    {
        perror("clock gettime");
    }

    // Launch Kernel
    kernel_sha256d<<<DimGrid, DimBlock>>>(d_ctx, d_nr, (void *) d_debug, d_hashes);
    
    // CPU Pthread execution
    for (j = 0; j < threadNum; j++)
    {
        threadDataArray[j].m_nonce = j * elementsPerThread;
        threadDataArray[j].m_ctx = &cpu_ctx;
        threadDataArray[j].m_nr = &cpu_nr;
        threadDataArray[j].m_length = elementsPerThread;
        int rc = pthread_create(&threads[j], NULL, KernelSHA256d, (void*)&threadDataArray[j]);
        if (rc)
        {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(-1);
        }
    }
    
 
    //Stop GPU timers
    hipEventRecord(gpu_stop,0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
     
    // GPU  Copy Nonce result back to host
    CUDA_SAFE_CALL(hipMemcpy((void *) &gpu_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));
   
    // GPU Syncrhonize
    hipDeviceSynchronize();



    // Pthread Synchronization
    for (i = 0; i < threadNum; i++)
    {
        int rc = pthread_join(threads[i], NULL);
        if (rc)
        {
            printf("joining error %d\n", rc);
            exit(-1);
        }
    }
    
    // Stop CPU Pthread Timer
    if(clock_gettime(CLOCK_REALTIME, &stop) == -1)
    {
        perror("clock gettime");
    }


    //Free memory on device
    CUDA_SAFE_CALL(hipFree(d_ctx));
    CUDA_SAFE_CALL(hipFree(d_nr));
    CUDA_SAFE_CALL(hipFree(d_debug));



    // CPU Results
    cpu_time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
    // printf("Total number of threads = %ld\n", threadNum);
    printf("\nTotal number of tested hashes = %ld\n", hashes);
    printf("Execution time = %f nano sec\n", cpu_time);
    printf("Hashrate = %f hashes/second\n", hashes / (cpu_time*1e-9));
    
    // GPU Results
    num_hashes = BDIMX;
    num_hashes *= GRIMX*GRIMY;
    printf("\nTested %lld hashes\n", num_hashes);
    printf("GPU execution time: %f ms\n", gpu_time);
    printf("Hashrate: %.2f H/s\n", num_hashes/(gpu_time*1e-3));
   
    if (cpu_nr.nonce_found)
    {
        printf("\nNonce found in CPU! %.8x\n", cpu_nr.nonce);
    }
    else
    {
        printf("\nNonce not found :(\n");
    }
    
    if (gpu_nr.nonce_found)
    {
        printf("\nNonce found in GPU! %.8x\n", gpu_nr.nonce);
    }
    else
    {
        printf("\nNonce not found :(\n");
    }

    return 0;
}

//Declare SHA-256 constants
__constant__ uint32_t k_[64] = {                                              0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
    0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
        0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
            0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
                0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
                    0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
                        0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
                            0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};


#define NONCE_VAL (gridDim.x*blockDim.x*blockIdx.y + blockDim.x*blockIdx.x + threadIdx.x)



__global__ void kernel_sha256d(SHA256_CTX *ctx, Nonce_result *nr, void *debug, unsigned int *hash_limit)
{

    if(nr->nonce_found )
    {
        // printf("return: nonce found\n");
        return;
    }
    unsigned int m[64];
    unsigned int hash[8];
    unsigned int a,b,c,d,e,f,g,h,t1,t2;
    int i, j;
    unsigned int nonce = NONCE_VAL;

    // Heterogenous part
    if (nonce < *hash_limit)
    {
        // printf("return: nonce tested in CPU\n");
        return;
    }

    // printf("current nonce Value: %ld\n", nonce );
    //Compute SHA-256 Message Schedule
    unsigned int *le_data = (unsigned int *) ctx->data;
    for(i=0; i<16; i++)
    {
    m[i] = le_data[i];
    }
    
    //Replace the nonce
    m[3] = nonce;
    for ( ; i < 64; ++i)
    {
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
    }
    
    //Copy Initial Values into registers
    a = ctx->state[0];
    b = ctx->state[1];
    c = ctx->state[2];
    d = ctx->state[3];
    e = ctx->state[4];
    f = ctx->state[5];
    g = ctx->state[6];
    h = ctx->state[7];

    //This is a large multiline macro for the SHA256 compression rounds
    SHA256_COMPRESS_8X_GPU
    
    //Prepare input for next SHA-256
    m[0] = a + ctx->state[0];
    m[1] = b + ctx->state[1];
    m[2] = c + ctx->state[2];
    m[3] = d + ctx->state[3];
    m[4] = e + ctx->state[4];
    m[5] = f + ctx->state[5];
    m[6] = g + ctx->state[6];
    m[7] = h + ctx->state[7];
    
    //Pad the input
    m[8] = 0x80000000;
    for(i=9; i<15; i++)
    {
    m[i] = 0x00;
    }
    m[15] = 0x00000100;    //Write out l=256
    for (i=16 ; i < 64; ++i)
    {
    m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
    }

    //Initialize the SHA-256 registers
    a = 0x6a09e667;
    b = 0xbb67ae85;
    c = 0x3c6ef372;
    d = 0xa54ff53a;
    e = 0x510e527f;
    f = 0x9b05688c;
    g = 0x1f83d9ab;
    h = 0x5be0cd19;
    
    SHA256_COMPRESS_1X_GPU

    hash[0] = ENDIAN_SWAP_32(a + 0x6a09e667);
    hash[1] = ENDIAN_SWAP_32(b + 0xbb67ae85);
    hash[2] = ENDIAN_SWAP_32(c + 0x3c6ef372);
    hash[3] = ENDIAN_SWAP_32(d + 0xa54ff53a);
    hash[4] = ENDIAN_SWAP_32(e + 0x510e527f);
    hash[5] = ENDIAN_SWAP_32(f + 0x9b05688c);
    hash[6] = ENDIAN_SWAP_32(g + 0x1f83d9ab);
    hash[7] = ENDIAN_SWAP_32(h + 0x5be0cd19);

    #ifdef VERIFY_HASH
    unsigned int *ref_hash = (unsigned int *) debug;
    for(i=0; i<8; i++)
    {
    //  cuPrintf("%.8x, %.8x\n", hash[i], ref_hash[i]);
    }
    #endif

    unsigned char *hhh = (unsigned char *) hash;
    i=0;
    while(hhh[i] == ctx->difficulty[i])
    {
    i++;
    }

    if(hhh[i] < ctx->difficulty[i])
    {
    //Synchronization Issue
    //Kind of a hack but it really doesn't matter which nonce
    //is written to the output, they're all winners :)
    //Further it's unlikely to even find a nonce let alone 2

    nr->nonce_found = true;
    //The nonce here has the correct endianess,
    //but it must be stored in the block in little endian order
    nr->nonce = nonce;
    }
}
