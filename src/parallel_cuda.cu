#include "../inc/parallel_cuda.h"

#include <cstdio>
#include <cstdlib>
#include <stdbool.h>
#include <stdint.h>

int main(int argc, char* argv[])
{
    int i, j;
    unsigned char* data = test_block;
    
    if (argc < 4)
    {
        printf("Wrong argument. Sample correct format: ./gpu_miner grid.x(8192) grid.y(8192) block.x(64)\n");
        return -1;
    }

    const unsigned int gridX = atoi(argv[1]);
    const unsigned int gridY = atoi(argv[2]);
    const unsigned int blockX = atoi(argv[3]);
    /*
        Host Side Preprocessing
        The goal here is to prepare and compute everything that will be shared by all threads.
    */
    
    //Initialize Cuda stuff
    dim3 DimGrid(gridX, gridY);
    #ifndef ITERATE_BLOCKS
    dim3 DimBlock(blockX, 1);
    #endif

    //Used to store a nonce if a block is mined
    Nonce_result h_nr;
    initialize_nonce_result(&h_nr);

    //Compute the shared portion of the SHA-256d calculation
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, (unsigned char*) data, 80);    //ctx.state contains a-h
    sha256_pad(&ctx);
    //Rearrange endianess of data to optimize device reads
    unsigned int* le_data = (unsigned int*)ctx.data;
    unsigned int le;
    for(i = 0, j = 0; i < 16; i++, j += 4)
    {
        //Get the data out as big endian
        //Store it as little endian via x86
        //On the device side cast the pointer as int* and dereference it correctly
        le = (ctx.data[j] << 24) | (ctx.data[j + 1] << 16) | (ctx.data[j + 2] << 8) | (ctx.data[j + 3]);
        le_data[i] = le;
    }

    //Decodes and stores the difficulty in a 32-byte array for convenience
    customize_difficulty(ctx.difficulty, 2);

    //Data buffer for sending debug information to/from the GPU
    unsigned char debug[32];
    unsigned char* d_debug;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_debug, 32 * sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpy(d_debug, (void *) &debug, 32 * sizeof(unsigned char), hipMemcpyHostToDevice));

    //Allocate space on Global Memory
    SHA256_CTX* d_ctx;
    Nonce_result* d_nr;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_ctx, sizeof(SHA256_CTX)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));

    /*
        Kernel Execution
        Measure and launch the kernel and start mining
    */
    //Copy data to device
    CUDA_SAFE_CALL(hipMemcpy(d_ctx, (void *) &ctx, sizeof(SHA256_CTX), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_nr, (void *) &h_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));

    float elapsed_gpu;
    long long int num_hashes;
    //Start timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Launch Kernel
    kernel_sha256d<<<DimGrid, DimBlock>>>(d_ctx, d_nr, (void *) d_debug);

    //Stop timers
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Copy nonce result back to host
    CUDA_SAFE_CALL(hipMemcpy((void *) &h_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));

    /*    
        Post Processing
        Check the results of mining and print out debug information
    */

    //Cuda Printf output
    hipDeviceSynchronize();

    //Free memory on device
    CUDA_SAFE_CALL(hipFree(d_ctx));
    CUDA_SAFE_CALL(hipFree(d_nr));
    CUDA_SAFE_CALL(hipFree(d_debug));

    //Output the results
    if(h_nr.nonce_found)
    {
        printf("Nonce found! %.8x\n", h_nr.nonce);
        compute_and_print_hash(data, h_nr.nonce);
    }
    else
    {
        printf("Nonce not found :(\n");
    }
    
    num_hashes = blockX;
    num_hashes *= gridX * gridY;
    printf("Tested %lld hashes\n", num_hashes);
    printf("GPU execution time: %f ms\n", elapsed_gpu);
    printf("Hashrate: %.2f H/s\n", num_hashes/(elapsed_gpu * 1e-3));

    return 0;
}
